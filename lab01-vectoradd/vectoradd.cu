#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<assert.h>
#include<unistd.h>

#define NUMELEMENT 1E6

void vecADD(float* h_A, float* h_B, float* h_C, int n)
{
	for (int i = 0 ; i < n ; i++)
		h_C[i] = h_A[i] + h_B[i];
}

__global__ void vecADDKernel(float* d_A, float* d_B, float* d_C, int n)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (i < n)
		d_C[i] = d_A[i] + d_B[i];
}

double get_mesc (struct timespec t1, struct timespec t2){
	return ((t2.tv_sec-t1.tv_sec)*1E9 + (t2.tv_nsec - t1.tv_nsec))/1E3;
}

struct timespec t_start, t_end;

int main()
{
	float *h_A, *h_B, *h_C;
	
	h_A = (float *)malloc(NUMELEMENT*sizeof(float));
	h_B = (float *)malloc(NUMELEMENT*sizeof(float));
	h_C = (float *)malloc(NUMELEMENT*sizeof(float));

	srand(222);

	for (int i = 0 ; i < NUMELEMENT ; i++)
	{
		h_A[i] = (float)(rand()%1000)/1000;
		h_B[i] = (float)(rand()%1000)/1000;
		h_C[i] = 0.0;
	}

	float *d_A, *d_B, *d_C;
	hipMalloc((void**)&d_A,NUMELEMENT*sizeof(float));
	hipMalloc((void**)&d_B,NUMELEMENT*sizeof(float));
	hipMalloc((void**)&d_C,NUMELEMENT*sizeof(float));

	hipMemcpy(d_A,h_A,NUMELEMENT*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_B,h_B,NUMELEMENT*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_C,h_C,NUMELEMENT*sizeof(float),hipMemcpyHostToDevice);


	clock_gettime(CLOCK_REALTIME,&t_start);	
	vecADD(h_A,h_B,h_C,NUMELEMENT);
	clock_gettime(CLOCK_REALTIME,&t_end);	
	printf("CPU Time :: %.0f msec\n",get_mesc(t_start,t_end));


	clock_gettime(CLOCK_REALTIME,&t_start);	
	vecADDKernel<<<ceil(NUMELEMENT/64),64>>>(d_A,d_B,d_C,NUMELEMENT);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_REALTIME,&t_end);	
	printf("GPU Time :: %.0f msec\n",get_mesc(t_start,t_end));


	float* h_R = (float *)malloc(NUMELEMENT*sizeof(float));
	hipMemcpy(h_R,d_C,NUMELEMENT*sizeof(float),hipMemcpyDeviceToHost);

	for (int i = 0 ; i < NUMELEMENT ; i++)
		assert(h_R[i] == h_C[i]);

	free(h_A);
	free(h_B);
	free(h_C);
	free(h_R);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}

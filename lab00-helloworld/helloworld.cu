#include <hip/hip_runtime.h>
#include <stdio.h>

///A device kernel function, just print out "Hello World"
__global__
void helloworld()
{
	printf("Hello World\n");
}

int main(int argc, char** argv)
{
	///Launch 1 block , each block has 1 thread 
	printf("Launch 1 block , each block has 1 thread \n");
	helloworld<<<  1, 1 >>>();
	printf("\n");
	
	///Launch 1 block , each block has 3 threads
	// printf("Launch 1 block , each block has 3 threads\n");
	// helloworld<<<  /*TODO*/, 3 >>>();
	// printf("\n");
	
	///Launch 2 blocks, each block has 1 thread
	// printf("Launch 2 blocks, each block has 1 thread \n");
	// helloworld<<<  /*TODO*/, 3 >>>();
	// printf("\n");
	
	///Launch 2 blocks, each block has 3 threads
	// printf("Launch 2 blocks, each block has 3 threads\n");
	// helloworld<<<  /*TODO*/, /*TODO*/ >>>();
	// printf("\n");
	
	return 0;
}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void helloword()
{
	printf("Hello World\n")
}

int main(int argc, char** argv)
{
	//Launch 1 block , each block has 1 thread 
	printf("Launch 1 block , each block has 1 thread \n");
	helloword<<<  1, 1 >>>();
	printf("\n");
	
	//Launch 1 block , each block has 3 threads
	// printf("Launch 1 block , each block has 3 threads\n");
	// helloword<<<  /*TODO*/, 3 >>>();
	
	//Launch 2 blocks, each block has 1 thread
	// printf("Launch 2 blocks, each block has 1 thread \n");
	// helloword<<<  /*TODO*/, 3 >>>();
		
	//Launch 2 blocks, each block has 3 threads
	// printf("Launch 2 blocks, each block has 3 threads\n");
	// helloword<<<  /*TODO*/, /*TODO*/ >>>();
	
	return 0;
}
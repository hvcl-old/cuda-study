#include <hip/hip_runtime.h>
#include <stdio.h>

///A device kernel function, just print out "Hello World"
__global__
void helloworld()
{
	printf("Hello World\n");
}

int main(int argc, char** argv)
{
	/// Use the first GPU on the machine
	hipSetDevice(0);
	
	///Launch 1 block , each block has 1 thread 
	printf("Launch 1 block , each block has 1 thread \n");
	helloworld<<<  1, 1 >>>();
	printf("\n");
	
	///Launch 1 block , each block has 3 threads
	// printf("Launch 1 block , each block has 3 threads\n");
	// helloworld<<<  1, /*TODO*/ >>>();
	// printf("\n");
	
	///Launch 2 blocks, each block has 1 thread
	// printf("Launch 2 blocks, each block has 1 thread \n");
	// helloworld<<<  /*TODO*/, 1 >>>();
	// printf("\n");
	
	///Launch 2 blocks, each block has 3 threads
	// printf("Launch 2 blocks, each block has 3 threads\n");
	// helloworld<<<  /*TODO*/, /*TODO*/ >>>();
	// printf("\n");
	
	/// Why this? You can try to comment this one and see
	hipDeviceSynchronize(); 
	return 0;
}